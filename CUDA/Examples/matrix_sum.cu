#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 512
#define BLOCK_DIM 512

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void matrixAdd (int *a, int *b, int *c)
{
    int col = //FIXME;
    int row = //FIXME;
    int index = col + row * N;

    if (col < N && row < N) {
    if (/*FIXME*/) {
        c[index] = a[index] + b[index];
    }
}

int main()
{
    int wrapSize = 32;
    int device_id = 0;
    int i = 0, j = 0;
    int a[N][N], b[N][N], c[N][N];
    int *dev_a, *dev_b, *dev_c;
    int size = N * N * sizeof(int);

    // CUDA device properties variable
    hipDeviceProp_t prop;

    // Query GPU properties
    /* FIXME */
    printf("maxThreadsDim x,y,z = %d,%d,%d\n",
            prop.FIXME,
            prop.FIXME,
            prop.FIXME);
    printf("maxGridSize x,y,z = %d,%d,%d\n",
            prop.FIXME,
            prop.FIXME,
            prop.FIXME);
    printf("maxThreadsPerBlock = %d, maxThreadsPerMultiProcessor = %d, maxBlocksPerMultiProcessor = %d\n",
            prop.FIXME
            prop.FIXME,
            prop.FIXME);
    printf("reservedSharedMemPerBlock = %d, sharedMemPerBlock = %d\n",
            prop.FIXME,
            prop.FIXME);

    // initialize a and b with real values (NOT SHOWN)
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    for (i = 0; i < N; ++i) {
        for (j = 0; j < N; ++j) {
            a[i][j] = b [i][j] = 1;
        }
    }
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(/*FIXME,FIXME*/);
    dim3 dimGrid(/*FIXME,FIXME*/);

    printf("dimBlock.x = %d, dimBlock.y = %d, dimBlock.z = %d\n",
            dimBlock.x, dimBlock.y, dimBlock.z);
    printf("dimGrid.x = %d, dimGrid.y = %d, dimGrid.z = %d\n",
            dimGrid.x, dimGrid.y, dimGrid.z);

    matrixAdd<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);
    cudaCheckErrors("kernel launch failure");

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    for (i = 0; i < N; ++i) {
        for (j = 0; j < N; ++j) {
            if (c[i][j] != 2) {
                printf("Data validation error at location c[%d][%d]. Expected: 2, Actual: %d (%d, %d)\n",
                        i, j, c[i][j], a[i][j], b[i][j]);
                exit(-1);
            }
        }
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
