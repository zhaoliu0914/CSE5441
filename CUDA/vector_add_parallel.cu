#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main(int argc, char *argv[]) {
    int i = 0;
    /* Host copies of a, b, and c */
    int *a = NULL, *b = NULL, *c = NULL;
    /* Device copies of a, b, and c */
    int *d_a = NULL, *d_b = NULL, *d_c = NULL;
    /* Amount of data */
    long int size = 0;
    /* Number of elements in the array */
    long int N = 0;
    /* Print or not */
    int print = 0;

    if (argc <= 1) {
        fprintf(stderr, "This program expects one input - size of array\n");
        exit(1);
    }
    /* Read number of elements from command line */
    N = atoi(argv[1]);
    /* Compute the size */
    size = N * sizeof(int);
    /* Error check */
    if (size <= 0) {
        fprintf(stderr, "Size of array should be greater than 0\n");
        exit(1);
    }
    if (argc == 3) {
        /* Read print option from the command line */
        print = atoi(argv[2]);
    }

    /* Allocate space for host copies of a, b, and c */
    a = (int *) malloc(size);
    b = (int *) malloc(size);
    c = (int *) malloc(size);

    /* Allocate space for device copies of a, b, and c */
    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    /* Dummy input values */
    for (i = 0; i < N; ++i) {
        a[i] = b[i] = i;
        c[i] = 0;
    }

    /* Copy input to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    /* Launch kernel for addition with N blocks */
    add<<<N, 1>>>(d_a, d_b, d_c);

    /* Copy result back to the host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    /* Print result */
    if (print == 1) {
        for (i = 0; i < N; ++i) {
            printf("a[%3d] (%3d) + b[%3d] (%3d) = c[%3d] (%3d)\n",
                   i, a[i], i, b[i], i, c[i]);
        }
    }

    /* Cleanup */
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
